#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<math.h>
#define N 100
#define thread_count 5
__global__ void array_add(double *d_result, double *d_a,int n, int thread_cnt)
{
    int load=n/thread_count;
   __shared__ double s_sum[thread_count];
  s_sum[threadIdx.x]=0;
 
    if(threadIdx.x != thread_cnt-1)
    {
        for(int i = (threadIdx.x)*load;i<((threadIdx.x+1)*load); i++)
        {
            s_sum[threadIdx.x]+= d_a[i];
        }
       
    }
      
    if(threadIdx.x == thread_cnt-1)
    {
        
       for(int i = (threadIdx.x)*load;i<n; i++)
        {
            s_sum[threadIdx.x]+= d_a[i];
        }
    }
    __syncthreads();
    
     if(threadIdx.x == 0 )
    {
        *d_result = 0;
        for(long int i=0; i<thread_cnt; i++)
        {
            *d_result += s_sum[i];
        }
    }
  
 
} 
int main()
{
    
    double *a ,out; 
    double *d_a,*d_result;
    
    a   = (double*)malloc(sizeof(double) * N);
    hipMalloc((void**)&d_a, sizeof(double) * N);
   hipMalloc((void**)&d_result, sizeof(double));
  
    //out = (double*)malloc(sizeof(double));
    

   

    for(int i = 0; i < N; i++)
    {
        a[i] = i;//pow(2,15)+rand()+0.13246549884;

    }

 hipMemcpy(d_a, a, sizeof(double)*N,hipMemcpyHostToDevice);
  
  
  array_add<<<1,thread_count>>>(d_result, d_a,N,thread_count);
 
     hipMemcpy(&out, d_result, sizeof(double), hipMemcpyDeviceToHost);
      //for(int i = 0; i < N; i++)
       printf(" sum=   %lf \n",out);
   
}
