#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<math.h>
#define N 10
#define thread_count 5

__global__ void dot_pro(double *d_result, double *d_a,double *d_b,int n, int thread_cnt)
{
    int load=n/thread_count;
   __shared__ double s_dsum[N];
  //s_dsum[threadIdx.x]=0;
 
    if(threadIdx.x != thread_cnt-1)
    {
        for(int i = (threadIdx.x)*load;i<((threadIdx.x+1)*load); i++)
        {
            s_dsum[i]= d_a[i]*d_b[i];
        }
       
    }
      
    if(threadIdx.x == thread_cnt-1)
    {
        
       for(int i = (threadIdx.x)*load;i<n; i++)
        {
            s_dsum[i]= d_a[i]*d_b[i];
        }
    }
    __syncthreads();
    
     if(threadIdx.x == 0 )
    {
        *d_result = 0;
        for( int i=0; i<N; i++)
        {
            *d_result += s_dsum[i];
        }
    }
  
 
} 
int main()
{
    
    double *a,*b ,out; 
    double *d_a,*d_b,*d_result;
    
    a   = (double*)malloc(sizeof(double) * N);
    b   = (double*)malloc(sizeof(double) * N);
    
    hipMalloc((void**)&d_a, sizeof(double) * N);
    hipMalloc((void**)&d_b, sizeof(double) * N);
    hipMalloc((void**)&d_result, sizeof(double));
  
    //out = (double*)malloc(sizeof(double));
    

   

    for(int i = 0; i < N; i++)
    {
        a[i] = i;//pow(2,15)+rand()+0.13246549884;
        b[i] = i;
    }

    hipMemcpy(d_a, a, sizeof(double)*N,hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double)*N,hipMemcpyHostToDevice);
  
  dot_pro<<<1,thread_count>>>(d_result, d_a,d_b,N,thread_count);
 
     hipMemcpy(&out, d_result, sizeof(double), hipMemcpyDeviceToHost);
      //for(int i = 0; i < N; i++)
       printf(" sum=   %lf \n",out);
   
}
