#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<math.h>
#define N 1000000
int thread_count = 24;
__global__ void vector_add(double *d_result, double *d_a, double *d_b, int n, int thread_cnt)
{
    int load=n/thread_cnt;
    if(threadIdx.x != thread_cnt-1)
    {
        for(int i = (threadIdx.x)*load;i<((threadIdx.x+1)*load); i++)
        {
            d_result[i] = d_a[i] + d_b[i];
        }
    }
    if(threadIdx.x == thread_cnt-1)
    {
        for(int i = (threadIdx.x)*load;i<n; i++)
        {
            d_result[i] = d_a[i] + d_b[i];
        }
    }
}

int main()
{
    double *a, *b, *out; 
    double *d_a,*d_b,*d_result;

    a   = (double*)malloc(sizeof(double) * N);
    hipMalloc((void**)&d_a, sizeof(double) * N);
    b   = (double*)malloc(sizeof(double) * N);
    hipMalloc((void**)&d_b, sizeof(double) * N);
    out = (double*)malloc(sizeof(double) * N);
    hipMalloc((void**)&d_result, sizeof(double) * N);

   

    for(int i = 0; i < N; i++)
    {
        a[i] = pow(2,15)+rand()+0.13246549884;;
      b[i] = pow(2,16)+rand()+(0.62477549884*i);
    }
 hipMemcpy(d_a, a, sizeof(double) * N, hipMemcpyHostToDevice);
   hipMemcpy(d_b, b, sizeof(double) * N, hipMemcpyHostToDevice);
  
  vector_add<<<1,thread_count>>>(d_result, d_a, d_b, N,thread_count);
 
     hipMemcpy(out, d_result, sizeof(double) * N, hipMemcpyDeviceToHost);
      for(int i = 0; i < N; i++)
          printf(" i= %d    %lf \n",i,out[i]);
}
