#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include<sys/time.h>
#define M 10
#define N 10
#define num_threads 4


__global__ void add(double *a, double *b, double *c) 
{
   int B=5;
    int elementsReceived = N/num_threads;
    int mini,minik;
    double r;

    if(num_threads != 1)
    {
        int start_index,end_index;
        start_index=(threadIdx.x * elementsReceived);
        if(threadIdx.x==num_threads-1)
          end_index=N;//Num of rows
        else
          end_index=(threadIdx.x + 1)*elementsReceived;
          
        for(int jj=0; jj<N; jj=jj+B)
        for(int kk=0; kk<N; kk=kk+B)
        for(int i=start_index; i<end_index; i++)
        {
             if (jj+B<N)
                mini=jj+B;
            else
                mini=N;
            for(int j=jj;j<mini;j++)
            {
                r=0;
                 if (kk+B<N)
                   minik=kk+B;
                else
                   minik=N;
                for(int k=kk;k<minik;k++)
                  r+=a[i*N+k]* b[k*N+j];   
                
                c[i*N+j]+=r;   
              
            }
            
        }
    }
    else
    {
        for(int jj=0; jj<N; jj=jj+B)
        for(int kk=0; kk<N; kk=kk+B)
        for(int i=0; i<N; i++)
        {
            if (jj+B<N)
                mini=jj+B;
            else
                mini=N;
            for(int j=jj;j<mini;j++)
            {
                r=0;
                  if (kk+B<N)
                   minik=kk+B;
                else
                   minik=N;
             
                for(int k=kk;k<minik;k++)
                    r+=a[i*N+k]* b[k*N+j];
                
                c[i*N+j]+=r;     
            }
        }
    }
  
}

int main(void) {
  double *a, *b, *c;   // host copies of a, b, c 
  double *d_a, *d_b, *d_c;  // device copies of a, b, c 
  int size = sizeof(double)*N*N;
 
  double start,end;
  
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a=(double *)malloc(size);
  b=(double *)malloc(size);
  c=(double *)malloc(size);

  // Setup input values
  for(int i = 0; i < N; i++)
  {
      for(int j=0;j<N;j++)
      {
         a[i+j*N] = 1;//pow(2,15)+rand()+0.13246549884;;
         b[i+j*N] = 1;//pow(2,16)+rand()+(0.06277549884*j); 
      }   
  }
 
  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); 
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice); 

  // Launch add() kernel on GPU
  add<<<1,num_threads>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
for(int i = 0; i < N; i++)
      for(int j=0;j<N;j++)
             printf("%lf   \n",c[i*N+j]);
  
  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
 // printf("%lf",end-start);
  return 0;
 } 
