#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>

#define M 1000
#define N 1000
#define num_threads 1

__global__ void add(double *d_a, double *d_b, double *d_c) 
{
    int load = M/num_threads;
    if(num_threads != 1)
    {
        int start_index,end_index;
        start_index=(threadIdx.x * load);
        if(threadIdx.x==num_threads-1)
          end_index=M;
        else
          end_index=(threadIdx.x + 1)*load;
        for(int i=start_index; i<end_index; i++)
        {
            for(int j=0;j<N;j++)
            {
                for(int k=0;k<N;k++)
                {
                  d_c[i+j*M]=d_c[i+j*M]+d_a[i+k*M]* d_b[k+j*N];   
                }
            }
            
        }
    }
    else
    {
        for(int i=0; i<M; i++)
        {
            for(int j=0;j<N;j++)
            {
                for(int k=0;k<N;k++)
                {
                  d_c[i+j*M]=d_c[i+j*M]+d_a[i+k*M]* d_b[k+j*N];   
                }
            }
        }
    }
  
}

int main(void) {
  double *a, *b, *c;   
  double *d_a, *d_b, *d_c;
  int size = sizeof(double)*M*N;
 
 

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a=(double *)malloc(size);
  b=(double *)malloc(size);
  c=(double *)malloc(size);


  for(int i = 0; i < M; i++)
  {
      for(int j=0;j<N;j++)
      {
         a[i+j*M] = pow(2,15)+rand()+0.13246549884;;
         b[i+j*M] = pow(2,16)+rand()+(0.06277549884*j); 
      }   
  }
 

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); 
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice); 


  add<<<1,num_threads>>>(d_a, d_b, d_c);


  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
 } 
