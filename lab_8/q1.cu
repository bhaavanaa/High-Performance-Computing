#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include<sys/time.h>
#define M 10
#define N 10
#define num_threads 2


__global__ void add(double *d_a, double *d_b, double *d_c) 
{
    int load = M/num_threads;
    if(num_threads != 1)
    {
        int start_index,end_index;
        start_index=(threadIdx.x * load);
        if(threadIdx.x==num_threads-1)
          end_index=start_index+N-start_index;
        else
          end_index=(threadIdx.x + 1)*load;
        for(int i=start_index; i<end_index; i++)
        {
            for(int j=0;j<N;j++)
            {
              d_c[i*N+j]=d_a[i*N+j]+ d_b[i*N+j];  
            }
            
        }
    }
    else
    {
        for(int i=0; i<M; i++)
        {
            for(int j=0;j<N;j++)
            d_c[i*N+j]=d_a[i*N+j]+ d_b[i*N+j];
        }
    }
  
}

int main(void) {
  double *a, *b, *c;   
  double *d_a, *d_b, *d_c;
  int size = sizeof(double)*M*N;

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a=(double *)malloc(size);
  b=(double *)malloc(size);
  c=(double *)malloc(size);


  for(int i = 0; i < M; i++)
  {
      for(int j=0;j<N;j++)
      {
         a[i*N+j] = pow(2,15)+rand()+0.13246549884;;
         b[i*N+j] = pow(2,16)+rand()+(0.12345678904*j); 
      }
      
  }


  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); 
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice); 


  add<<<1,num_threads>>>(d_a, d_b, d_c);


  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

   /*for(int i = 0; i < M; i++)
      for(int j=0;j<N;j++)
        printf("%lf",c[i*N+j]);*/

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
 } 
